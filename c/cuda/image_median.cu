#include <opencv2/opencv.hpp>

#include <chrono>
#include <string>
#include "common.h"
#include "median.h"
#define KERNAL_SIZE 5
#define KERNAL_TOTAL_SIZE KERNAL_SIZE * KERNAL_SIZE

using namespace std;
using namespace std::chrono;

int main(int argc, char *argv[]) {

    try {
        std::cout << "Start: Program" << std::endl;

        string image_name;
        if(argc > 1)
            image_name = argv[1];
        else
            image_name = "sp_img_gray_noise_heavy.png";

        cv::Mat image = cv::imread("resources/" + image_name, cv::IMREAD_COLOR);

        int N_Channels = 3; // Number of Channels
        int rows = image.rows;
        int cols = image.cols;

        int gpu_device = -1;
        hipGetDevice(&gpu_device);

        cv::Mat frame, filteredFrame;

        cv::Mat channels[N_Channels], outputChannels[N_Channels];

        // GPU device source and destination matrices
        u_int8_t *h_channels[N_Channels], *h_outputChannels[N_Channels];
        u_int8_t *d_channels[N_Channels], *d_outputChannels[N_Channels];

        // // Save the frame before filtering
        std::string filenameBefore = "before.jpg" ;
        cv::imwrite(filenameBefore, image);

        cv::split(image, channels);

        std::cout << "Start: copying channels" << std::endl;

        auto start_cp = high_resolution_clock::now();

        for (int c = 0; c < N_Channels; c++) {
            hipHostMalloc(&h_channels[c], sizeof(u_int8_t) * rows * cols);
            CHECK_LAST_CUDA_ERROR();
            hipHostMalloc(&h_outputChannels[c], sizeof(u_int8_t) * rows * cols);
            CHECK_LAST_CUDA_ERROR();

            hipMemcpy(h_channels[c], channels[c].data, sizeof(u_int8_t) * rows * cols, hipMemcpyHostToHost);
        }
        std::cout << "Start: copying to device" << std::endl;
        
        for (int c = 0; c < N_Channels; c++) {
            hipMalloc(&d_channels[c], sizeof(u_int8_t) * rows * cols);
            CHECK_LAST_CUDA_ERROR();
            hipMalloc(&d_outputChannels[c], sizeof(u_int8_t) * rows * cols);
            CHECK_LAST_CUDA_ERROR();

            hipMemcpyAsync(d_channels[c], h_channels[c], sizeof(u_int8_t) * rows * cols, hipMemcpyHostToDevice);
            CHECK_LAST_CUDA_ERROR();
        }
        hipDeviceSynchronize();

        std::cout << "Start: kernal" << std::endl;

        auto start_mf = high_resolution_clock::now();
        // Apply median filter to each channel
        for (int i = 0; i < N_Channels; i++) {
            median_filter_driver(d_channels[i], d_outputChannels[i], rows, cols);
        }
        hipDeviceSynchronize();
        CHECK_LAST_CUDA_ERROR();
        auto end_mf = high_resolution_clock::now();

        for (int i = 0; i < N_Channels; i++) {
            outputChannels[i] = cv::Mat(rows, cols, CV_8UC1);
            hipMemcpyAsync(h_outputChannels[i], d_outputChannels[i], sizeof(u_int8_t) * rows*cols, hipMemcpyDeviceToHost);
            CHECK_LAST_CUDA_ERROR();
        }
        hipDeviceSynchronize();
        for (int i = 0; i < N_Channels; i++) {
            hipMemcpy(outputChannels[i].data, h_outputChannels[i], sizeof(u_int8_t) * rows*cols, hipMemcpyHostToHost);
            CHECK_LAST_CUDA_ERROR();
        }
        auto end_cp = high_resolution_clock::now();

        // Merge the channels back
        cv::merge(outputChannels, N_Channels, filteredFrame);

        // Save the frame after filtering
        std::string filenameAfter = "after.jpg";
        cv::imwrite(filenameAfter, filteredFrame);
        
        std::cout << "End : Start Image Saving" << std::endl;

        auto total_duration = duration_cast<microseconds>(end_cp - start_cp).count();
        auto filter_duration = duration_cast<microseconds>(end_mf - start_mf).count();

        std::cout << "total time    : " << total_duration << " us" << std::endl;
        std::cout << "filter time   : " << filter_duration << " us" << std::endl;
        std::cout << "mem time      : " << total_duration - filter_duration << " us" << std::endl;
        for (int c = 0; c < N_Channels; c++) {
            hipFree(d_channels[c]);
            CHECK_LAST_CUDA_ERROR();
            hipFree(d_outputChannels[c]);
            CHECK_LAST_CUDA_ERROR();
            hipHostFree(h_channels[c]);
            CHECK_LAST_CUDA_ERROR();
            hipHostFree(h_outputChannels[c]);
            CHECK_LAST_CUDA_ERROR();
        }
    } catch(const cv::Exception& ex) {
        std::cout << "Error: " << ex.what() << std::endl;
    }

    return 0;
}